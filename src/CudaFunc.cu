#include "hip/hip_runtime.h"
#include "CudaFunc.cuh"
#include ""

__global__ void updatePositions(
    double *directions, double *positions, double *speeds, Vertex *vertices, int *indices, int *status, int* unit_class,
    int map_sizeX, int map_sizeY, int map_sizeZ,
    int time_slice, int unit_num
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= unit_num) return;  // 确保线程处理的索引在有效范围内
    if (status[i] == 0) {
        vertices[i].status = 0;
        return;
    }
    double direction = sqrt(
        pow(directions[i * 3], 2) +
        pow(directions[i * 3 + 1], 2) +
        pow(directions[i * 3 + 2], 2)
    );

    if (direction == 0) return;

    double posX_add  = speeds[i] * time_slice * directions[i * 3]     / direction; // 三个方向上的增量
    double posY_add  = speeds[i] * time_slice * directions[i * 3 + 1] / direction;
    double posZ_add  = speeds[i] * time_slice * directions[i * 3 + 2] / direction;
    double next_posX = positions[i * 3]     + posX_add;
    double next_posY = positions[i * 3 + 1] + posY_add;
    double next_posZ = positions[i * 3 + 2] + posZ_add;

    if (next_posX > map_sizeX || next_posX < 0) posX_add = -posX_add; // 根据地图范围修正增量
    if (next_posY > map_sizeY || next_posY < 0) posY_add = -posY_add;
    if (next_posZ > map_sizeZ || next_posZ < 0) posZ_add = -posZ_add;

    positions[i * 3]     = positions[i * 3]     + posX_add;
    positions[i * 3 + 1] = positions[i * 3 + 1] + posY_add;
    positions[i * 3 + 2] = positions[i * 3 + 2] + posZ_add;

    float scale = 2000.0;
    glm::vec3 pos((positions[i * 3] - 5000) / scale, (positions[i * 3 + 1] - 5000) / scale, positions[i * 3 + 2]);
    glm::vec3 col(1, 0, 0);
    int camp = (i < unit_num / 2) ? 0 : 1;

    // indices[i] = i;
    vertices[i] = Vertex(pos, col, status[i], camp);
}

__global__ void calculateDistances(
    double *positions, double *target_positions, double *distances, int unit_num
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < unit_num) { // 各个单位与目标之间的distances
        distances[i] = sqrt(pow(target_positions[i * 3]     - positions[i * 3],     2) 
                          + pow(target_positions[i * 3 + 1] - positions[i * 3 + 1], 2) 
                          + pow(target_positions[i * 3 + 2] - positions[i * 3 + 2], 2));
    } else if (i < unit_num * 2) { // 每个单位与camp0阵营的基站（索引为0）的距离
        distances[i] = sqrt(pow(positions[0] - positions[(i - unit_num) * 3],     2) 
                          + pow(positions[1] - positions[(i - unit_num) * 3 + 1], 2) 
                          + pow(positions[2] - positions[(i - unit_num) * 3 + 2], 2));
    } else if (i < unit_num * 3) { // 每个单位与camp1阵营的基站（索引为unit_num / 2）的距离
        distances[i] = sqrt(pow(positions[(unit_num / 2) * 3]     - positions[(i - unit_num * 2) * 3],     2) 
                          + pow(positions[(unit_num / 2) * 3 + 1] - positions[(i - unit_num * 2) * 3 + 1], 2) 
                          + pow(positions[(unit_num / 2) * 3 + 2] - positions[(i - unit_num * 2) * 3 + 2], 2));
    }
}

__global__ void unitUpdate(
    double *positions, double *target_positions, double *directions, double *distances,
    int *target_ids, double *init_positions, int *weapon_nums, int *status, double *attack_radius, int *unit_class,
    int unit_num
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= unit_num ) return;
    if (status[i] == 0) return;
    if (unit_class[i] == 0) return;

    hiprandStateXORWOW_t rand_state;
    hiprand_init(0,i,0,&rand_state);

    if (target_positions[i * 3] == 0 && target_positions[i * 3 + 1] == 0 && target_positions[i * 3 + 2] == 0) { // 如果没有目标，随机前进
        directions[i * 3]           = hiprand_uniform(&rand_state) * 10;          // 更新运动方向
        directions[i * 3 + 1]       = hiprand_uniform(&rand_state) * 10;
        if (unit_class[i] == 1 || unit_class[i] == 2) directions[i * 3 + 2] = hiprand_uniform(&rand_state) * 10; // 飞行单位才有z轴速度
        else directions[i * 3 + 2] = 0;
    } else if (distances[i] < attack_radius[i]) { // 有目标，距离足够
        if (weapon_nums[i] > 0) weapon_nums[i] -= 1;        // 攻击
        if (target_ids[i] != 0) {   // 更新攻击目标status
            status[target_ids[i]] = 0;  
            target_ids[i] = 0;
        }
        if (unit_class[i] == 2) status[i] = 0;              // 如果是Missle，自己也更新status
        target_positions[i * 3]     = 0;                    // 更新目标位置
        target_positions[i * 3 + 1] = 0;
        target_positions[i * 3  +2] = 0;
        if (unit_class[i] == 3 || unit_class[i] == 4) { // 地面单位需要返回初始位置
            directions[i * 3]       = init_positions[i * 3]     - positions[i * 3];       // 更新运动方向
            directions[i * 3 + 1]   = init_positions[i * 3 + 1] - positions[i * 3 + 1];
            directions[i * 3 + 2]   = init_positions[i * 3 + 2] - positions[i * 3 + 2];
        } else {
            directions[i * 3]       = hiprand_uniform(&rand_state) * 10;          // 更新运动方向
            directions[i * 3 + 1]   = hiprand_uniform(&rand_state) * 10;
            directions[i * 3 + 2]   = hiprand_uniform(&rand_state) * 10;
        }
    } else { // 距离不够，继续前进
        directions[i * 3]           = target_positions[i * 3]     - positions[i * 3];
        directions[i * 3 + 1]       = target_positions[i * 3 + 1] - positions[i * 3 + 1];
        directions[i * 3 + 2]       = target_positions[i * 3 + 2] - positions[i * 3 + 2];
    }
}

int cudaUpdatePositions(
    double *directions, double *positions, double *speeds, Vertex *vertices, int *indices,
    double *g_directions, double *g_positions, double *g_speeds, Vertex *g_vertices, int *g_indices, int *g_status, int* g_unit_class,
    int map_sizeX, int map_sizeY, int map_sizeZ,
    int time_slice, int unit_num
) {
    hipError_t err;

    // 定义线程块大小和网格大小
    int blockSize = 1024; // 根据你的 GPU 调整
    int gridSize = (unit_num + blockSize - 1) / blockSize;

    // 调用内核
    updatePositions<<<gridSize, blockSize>>>(g_directions, g_positions, g_speeds, g_vertices, g_indices, g_status, g_unit_class, map_sizeX, map_sizeY, map_sizeZ, time_slice, unit_num);

    // 错误检查
    hipDeviceSynchronize(); // 等待 CUDA 设备完成所有先前排队的工作
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    hipMemcpy(positions, g_positions,  sizeof(double) * 3 * unit_num, hipMemcpyDeviceToHost);
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Mem Error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    hipMemcpy(vertices, g_vertices,    sizeof(Vertex) * unit_num, hipMemcpyDeviceToHost);
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Mem Error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    hipMemcpy(indices, g_indices,      sizeof(int) * unit_num, hipMemcpyDeviceToHost);
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Mem D2H Error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    return 1;
}

int cudaCalculateDistances(
    double *positions, double *target_positions, double *distances,
    double *g_positions, double *g_target_positions, double *g_distances,
    int unit_num
) {
    hipError_t err;

    int blockSize = 1024; // 根据你的 GPU 调整
    int gridSize = (unit_num * 3 + blockSize - 1) / blockSize;

    // 调用内核
    calculateDistances<<<gridSize, blockSize>>>(g_positions, g_target_positions, g_distances, unit_num);

    // 错误检查
    hipDeviceSynchronize(); // 等待 CUDA 设备完成所有先前排队的工作
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    hipMemcpy(distances, g_distances, sizeof(double) * 3 * unit_num, hipMemcpyDeviceToHost);
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Mem D2H Error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    return 1;
}

int cudaUnitUpdate(
    double *g_positions, double *g_target_positions, double *g_directions, double *g_distances,
    int *g_target_ids, double *g_init_positions, int *g_weapon_nums, int *g_status, double *g_attack_radius, int *g_unit_class,
    int *target_ids, double *directions, int *weapon_nums, int *status, double *target_positions,
    int unit_num
) {
    hipError_t err;

    hipMemcpy(g_target_ids, target_ids, sizeof(int) * unit_num, hipMemcpyHostToDevice); // 基站的update会更新target_ids
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Mem Error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    hipMemcpy(g_target_positions, target_positions, sizeof(double) * 3 * unit_num, hipMemcpyHostToDevice); // 基站的update会更新target_positions
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Mem Error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    int blockSize = 1024; // 根据你的 GPU 调整
    int gridSize = (unit_num + blockSize - 1) / blockSize;

    // 调用内核
    unitUpdate<<<gridSize, blockSize>>>(g_positions, g_target_positions, g_directions, g_distances, 
                                        g_target_ids, g_init_positions, g_weapon_nums, g_status, g_attack_radius, g_unit_class,
                                        unit_num);

    // 错误检查
    hipDeviceSynchronize(); // 等待 CUDA 设备完成所有先前排队的工作
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    hipMemcpy(directions, g_directions, sizeof(double) * 3 * unit_num, hipMemcpyDeviceToHost);
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Mem D2H Error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    hipMemcpy(status, g_status, sizeof(int) * unit_num, hipMemcpyDeviceToHost);
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Mem D2H Error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    hipMemcpy(target_ids, g_target_ids, sizeof(int) * unit_num, hipMemcpyDeviceToHost);
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Mem Error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    hipMemcpy(target_positions, g_target_positions, sizeof(double) * 3 * unit_num, hipMemcpyDeviceToHost);
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Mem Error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    return 1;
}