#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>
#include <cstdlib>
#include <cmath>

__global__ void updatePositions(
    double *directions, double *positions, double *speeds,
    double map_sizeX, double map_sizeY, double map_sizeZ,
    double time_slice
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < 3000) { // 确保线程处理的索引在有效范围内
        double direction = sqrt(
            pow(directions[i * 3], 2) +
            pow(directions[i * 3 + 1], 2) +
            pow(directions[i * 3 + 2], 2)
        );
        double next_posX = positions[i * 3] + speeds[i] * time_slice * directions[i * 3] / direction;
        double next_posY = positions[i * 3 + 1] + speeds[i] * time_slice * directions[i * 3 + 1] / direction;
        double next_posZ = positions[i * 3 + 2] + speeds[i] * time_slice * directions[i * 3 + 2] / direction;

        if (next_posX > map_sizeX || next_posX < 0) directions[i * 3] = -directions[i * 3];
        if (next_posY > map_sizeY || next_posY < 0) directions[i * 3 + 1] = -directions[i * 3 + 1];
        if (next_posZ > map_sizeZ || next_posZ < 0) directions[i * 3 + 2] = -directions[i * 3 + 2];

        positions[i * 3] = next_posX; // 直接赋值计算后的位置
        positions[i * 3 + 1] = next_posY;
        positions[i * 3 + 2] = next_posZ;
    }
}

int main() {
    const int total_count = 3000;
    double *directions, *positions, *speeds;
    double map_sizeX = 100.0; // 假设地图尺寸为 100x100x100
    double map_sizeY = 100.0;
    double map_sizeZ = 100.0;
    double time_slice = 0.1; // 假设时间切片为 0.1

    // 分配 GPU 内存
    hipMalloc(&directions, total_count * 3 * sizeof(double));
    hipMalloc(&positions, total_count * 3 * sizeof(double));
    hipMalloc(&speeds, total_count * sizeof(double));

    // 初始化随机数据
    double *h_directions = new double[total_count * 3];
    double *h_positions = new double[total_count * 3];
    double *h_speeds = new double[total_count];

    srand(time(NULL)); // 设置随机数种子
    for (int i = 0; i < total_count * 3; ++i) {
        h_directions[i] = (rand() / (double)RAND_MAX) * 2 - 1; // -1 到 1 之间的随机数
        h_positions[i] = (rand() / (double)RAND_MAX) * map_sizeX; // 0 到 map_sizeX 之间的随机数
    }
    for (int i = 0; i < total_count; ++i) {
        h_speeds[i] = (rand() / (double)RAND_MAX) * 0.5 + 0.1; // 0.1 到 0.6 之间的随机速度
    }

    // 从主机复制数据到 GPU
    hipMemcpy(directions, h_directions, total_count * 3 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(positions, h_positions, total_count * 3 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(speeds, h_speeds, total_count * sizeof(double), hipMemcpyHostToDevice);

    // 定义线程块大小和网格大小
    int blockSize = 256; // 根据你的 GPU 调整
    int gridSize = (total_count + blockSize - 1) / blockSize;

    // 记录开始时间
    clock_t start_time = clock();

    for (int i = 0; i < 3; ++i) { // 执行三次内核调用
        // 调用内核
        updatePositions<<<gridSize, blockSize>>>(directions, positions, speeds, map_sizeX, map_sizeY, map_sizeZ, time_slice);

        // 等待设备完成所有工作
        hipDeviceSynchronize();

        // 检查是否有错误发生
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
            return -1;
        }
        hipMemcpy(directions, h_directions, total_count * 3 * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(positions, h_positions, total_count * 3 * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(speeds, h_speeds, total_count * sizeof(double), hipMemcpyHostToDevice);
    }

    // 记录结束时间
    clock_t end_time = clock();

    // 计算总时间（时钟周期数转换为秒）
    double elapsed_time = static_cast<double>(end_time - start_time) / CLOCKS_PER_SEC;

    // 打印总时间
    std::cout << "Elapsed time: " << elapsed_time << " seconds\n";
    // // 调用内核
    // updatePositions<<<gridSize, blockSize>>>(directions, positions, speeds, map_sizeX, map_sizeY, map_sizeZ, time_slice);

    // // 错误检查
    // cudaDeviceSynchronize(); // 等待 CUDA 设备完成所有先前排队的工作
    // cudaError_t err = cudaGetLastError();
    // if (err != cudaSuccess) {
    //     std::cerr << "CUDA Error: " << cudaGetErrorString(err) << std::endl;
    //     return -1;
    // }

    // 复制结果回主机内存（如果需要）
    double *h_positions_copy = new double[total_count * 3];
    hipMemcpy(h_positions_copy, positions, total_count * 3 * sizeof(double), hipMemcpyDeviceToHost);

    // TODO: 验证结果

    // 释放 GPU 内存
    hipFree(directions);
    hipFree(positions);
    hipFree(speeds);

    // 释放主机内存
    delete[] h_directions;
    delete[] h_positions;
    delete[] h_speeds;
    delete[] h_positions_copy;

    return 0;
}